#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"


__device__ unsigned int Amodprod(unsigned int a, unsigned int b, unsigned int p)
  {
    unsigned int za = a;
    unsigned int ab = 0;
    while(b>0)
    {
      if(b%2 == 1) ab = (ab + za) % p;
      za = (2 * za) % p;
      b /= 2;
    }
  return ab;
  }
__device__ unsigned int AmodExp(unsigned int a, unsigned int b, unsigned int p)
  {
    unsigned int z = a;
    unsigned int aExpb = 1;
    while (b>0)
    {
      if(b%2==1) aExpb = Amodprod(aExpb, z, p);
      z = Amodprod(z,z,p);
      b /= 2;
    }
  return aExpb;
  }
__global__ void findSecretKey(unsigned int N, unsigned int p, unsigned int g, unsigned int h,unsigned int *d_a)
  {
  int threadId = threadIdx.x;
  int blockId = blockIdx.x;
  int Nblock = blockDim.x;
  int id = threadId + blockId*Nblock;
      if(id<N)
        {
          if(AmodExp(g,id,p) ==h)
            {
                *d_a = id;
            }
        }
   }

int main (int argc, char **argv) {

  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h;
  unsigned int x;
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */

  FILE* file1 = fopen("public_key.txt","r");
  FILE* file2 = fopen("message.txt", "r");
  unsigned int n1;
  unsigned int n2;
  fscanf(file1, "%d", &n1);
  fscanf(file2, "%d", &n2);
  unsigned int *data1 = (unsigned int *) malloc(n1*sizeof(unsigned int));
  unsigned int *Zmessage = (unsigned int *) malloc(n2*sizeof(unsigned int));
  unsigned int *a = (unsigned int *) malloc(n2*sizeof(unsigned int));
  //read in first file
  for(unsigned int m1=0; m1<n1; m1++)
  {
    fscanf(file1, "%u", data1+m1);
  }
  fclose(file1);
  n = n1;
  p = data1[0];
  g = data1[1];
  h = data1[2];
  printf("n is %d\n", n);
  printf("p is %d\n", p);
  printf("g is %d\n", g);
  printf("h is %d\n", h);
  printf("x is %d\n", x);
  //read in second file
  for(unsigned int m2=0; m2<n2; m2++)
  {
    fscanf(file2, "%u  %u", Zmessage+m2, a+m2);
  }
  int numOfCypher = 0;
  numOfCypher = n2;
 // printf("testing here 1\n");
  fclose(file2);
  free(data1);
 // printf("testing here 2\n");
  //make cuda kernal
  //device arrays
  int Nthreads = 32;
  dim3 B(Nthreads,1,1);
  dim3 G(((p+Nthreads-1))/Nthreads,1,1);
 // printf("testing here 3\n");
  unsigned int *d_a, *h_a;
  hipMalloc(&d_a,sizeof(unsigned int));
  h_a = (unsigned int *) malloc(sizeof(unsigned int));
  double startTime = clock();
 // printf("testing here 4\n");
  findSecretKey<<<G,B>>>(p-1,p,g,h,d_a);
  hipDeviceSynchronize();
 // printf("extra testing 100\n");
  hipMemcpy(h_a,d_a,sizeof(unsigned int),hipMemcpyDeviceToHost);
 // printf("extra test 150\n");
  x=*h_a;
  printf("x is: %d\n", x);
 // printf("extra testing 200\n");
  ElGamalDecrypt(Zmessage,a,numOfCypher,p,x);
 // printf("extra testing 300\n");
  free(h_a);
  hipFree(d_a);
  //printf("testing here 5\n");
  //done with cuda
  double endTime = clock();
  double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
  double work = (double) p;
  double throughput = work/totalTime;
  printf("Searching all keys took %g seconds, throughput was %g values per second. \n", totalTime, throughput);
  int bufferSize = 1024;
  unsigned char *message = (unsigned char *) malloc(bufferSize*sizeof(unsigned char));
  unsigned int charsPerInt = ((n-1)/8)*numOfCypher;
  convertZToString(Zmessage, numOfCypher, message, charsPerInt);
 // printf("testing here 6\n");
  printf("Decrypted message = \"%s\"\n", message);
  printf("\n");
  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */

  return 0;
}
