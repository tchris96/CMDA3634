#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"


__device__ Amodprod(unsigned int a, unsigned int b, unsigned int p)
  {
    unsigned int za = a;
    unsigned int ab = 0;
    while(b>0)
    {
      if(b%2 == 1) ab = (ab + za) % p;
      za = (2 * za) % p;
      b /= 2;
    }
  return ab;
  }
__device__ AmodExp(unsigned int a, unsigned int b, unsigned int p)
  {
    unsigned int z = a;
    unsigned int aExpb = 1;
    while (b>0)
    {
      if(b%2==1) aExpb = modprod(aExpb, z, p);
      z = modprod(z,z,p);
      b /= 2;
    }
  return aExpb;
  }
//__device__ AconvertZToString(unsigned int *Z, unsigned int Nints, unsigned char *string, unsigned int Nchars)
//  {
//    unsigned int charsPerInt = Nchars/Nints;
//    for (int i=0; i<Nints; i++){
//      string[i*charsPerInt + charsPerInt-1-n] = z%256;
//      z /=256;
//    }
//  }
//  string[Nints*charsPerInt] = '\0';
//}
//__device__ AElGamalDecrypt(unsigned int *m, unsigned int *a, unsigned int Nintsm unsigned int p, unsigned int x){
//  for(unsigned int i=0; i<Nints;i++)
//  {
//    unsigned int s = modExp(a[i],x,p);
//    unsigned int invS = modExp(s,p-2,p);
//    m[i] = modprod(m[i], invS,p);
//  }
//}
__device__ findSecretKey(unsigned int n, unsigned int p, unsigned int g, unsigned int h, unsigned int x)
  {
    if(x==0 || AmodExp(g,x,p)!=h)
    {
      printf("Finding the secret key...\n");
      double startTime = clock();
      for(unsigned int i=0; i<p-1;i++)
      {
        if(AmodExp(g,i+1,p) ==h)
        {
          printf("Secret key found! x = %u \n", i+1);
        }
      }
    }
   }


int main (int argc, char **argv) {

  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */

  FILE* file1 = fopen("public_key.txt","r");
  FILE* file2 = fopen("message.txt", "r");
  unsigned int n1;
  unsigned int n2;
  fscanf(file1, "%d", &n1);
  fscanf(file2, "%d", &n2);
  unsigned int *data1 = (int *) malloc(n1*sizeof(int));
  unsigned int *Zmessage = (int *) malloc(n2*sizeof(int));
  unsigned int *a = (int *) malloc(n2*sizeof(int));

  for(unsigned int m1=0; m1<n1; m1++)
  {
    fscanf(file1, "%u", data1+m1);
  }
  n = n1;
  p = data1[0];
  g = data1[1];
  h = data1[2];
  printf("n is %d\n", n);
  printf("p is %d\n", p);
  printf("g is %d\n", g);
  printf("h is %d\n", h);
  printf("x is %d\n", x);
  for(unsigned int m2=0; m2<n2; m2++)
  {
    fscanf(file2, "%u  %u", Zmessage+m2, a+m2);
  }
  int numOfCypher = 0;
  numOfCypher = n2;
  printf("%d\n", Zmessage[0]);
  printf("%d\n", a[0]); 
//  if(x==0 || AmodExp(g,x,p)!=h){
//  printf("Finding the secret key...\n");
//  double startTime = clock();  
//  for(unsigned int i = 0; i<p-1;i++){
//  if(AmodExp(g,i+1,p)==h){
//  printf("Secret key found! x = %u \n", i+1);
//  x=i+1;
//  }
//  }
  double endTime = clock();
  double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
  double work =(double) p;
  double throughput = work/totalTime;
  printf("Searching all keys took %g seconds, throughput was %g values per second. \n", totalTime,throughput);
  }
  fclose(file1);
  fclose(file2);
  free(data1);
  //make cuda kernal

  //device arrays
  int Nre = atoi(argv[1]);
  int Nim = atoi(argv[2]);
  int Nthreads = atoi(argv[3]);
  dim3 B(Nthreads,Nthreads,1);
  dim3 G((Nre+Nthreads-1)/Nthreads,(Nim+Nthreads-1)/Nthreads,1);
  double *d_a, *h_a;
  hipMalloc(&d_a,N*sizeof(double));
  h_a = (double *) malloc(N*sizeof(double));
  //calculate secret key with cuda
  findSecretKey<<<G,B>>>(n,p,g,h,x);
  hipDeviceSynchronize();
  hipMemcpy(h_a,d_a,Nre*Nim*sizeof(float),hipMemcpyDeviceToHost);
  ElGamalDecrypt(Zmessage,a,numOfCypher,p,x);
  free(h_a);
  hipFree(d_a);
  //done with cuda
  int bufferSize = 1024;
  unsigned char *message = (unsigned char *) malloc(bufferSize*sizeof(unsigned char));
  unsigned int charsPerInt = ((n-1)/8)*numOfCypher;
  convertZToString(Zmessage, numOfCypher, message, charsPerInt);
  printf("Decrypted message = \s\"\n", message);
  printf("\n");
  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */

  return 0;
}
