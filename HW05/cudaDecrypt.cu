#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"


__device__ Amodprod(unsigned int a, unsigned int b, unsigned int p)
  {
    unsigned int za = a;
    unsigned int ab = 0;
    while(b>0)
    {
      if(b%2 == 1) ab = (ab + za) % p;
      za = (2 * za) % p;
      b /= 2;
    }
  return ab;
  }
__device__ AmodExp(unsigned int a, unsigned int b, unsigned int p)
  {
    unsigned int z = a;
    unsigned int aExpb = 1;
    while (b>0)
    {
      if(b%2==1) aExpb = Amodprod(aExpb, z, p);
      z = modprod(z,z,p);
      b /= 2;
    }
  return aExpb;
  }
__global__ findSecretKey(unsigned int n, unsigned int p, unsigned int g, unsigned int h, unsigned int *x)
  {
  int threadId = threadIdx.x;
  int blockId = blockIdx.x;
  int Nblock = blockDim.x;
  int id = threadId + blockID*Nblock;
    if(x==0 || AmodExp(g,x,p)!=h)
      {
      if(id<pow(2,n))
        {
    if(AmodExp(g,i+1,p) ==h)
            {
               *x[0] = i+1;
            }
        }
      }
   }

int main (int argc, char **argv) {

  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */

  FILE* file1 = fopen("public_key.txt","r");
  FILE* file2 = fopen("message.txt", "r");
  unsigned int n1;
  unsigned int n2;
  fscanf(file1, "%d", &n1);
  fscanf(file2, "%d", &n2);
  unsigned int *data1 = (int *) malloc(n1*sizeof(int));
  unsigned int *Zmessage = (int *) malloc(n2*sizeof(int));
  unsigned int *a = (int *) malloc(n2*sizeof(int));
  //read in first file
  for(unsigned int m1=0; m1<n1; m1++)
  {
    fscanf(file1, "%u", data1+m1);
  }
  n = n1;
  p = data1[0];
  g = data1[1];
  h = data1[2];
  printf("n is %d\n", n);
  printf("p is %d\n", p);
  printf("g is %d\n", g);
  printf("h is %d\n", h);
  printf("x is %d\n", x);
  //read in second file
  for(unsigned int m2=0; m2<n2; m2++)
  {
    fscanf(file2, "%u  %u", Zmessage+m2, a+m2);
  }
  int numOfCypher = 0;
  numOfCypher = n2;
  printf("%d\n", Zmessage[0]);
  printf("%d\n", a[0]); 
//  double endTime = clock();
//  double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
//  double work =(double) p;
//  double throughput = work/totalTime;
//  printf("Searching all keys took %g seconds, throughput was %g values per second. \n", totalTime,throughput);
  }
  fclose(file1);
  fclose(file2);
  free(data1);
  //make cuda kernal
  //device arrays
//  int Nre = atoi(argv[1]);
//  int Nim = atoi(argv[2]);
  int Nthreads = 32;
  dim3 B(Nthreads,1,1);
  dim3 G(((p+Nthreads-1))/Nthreads,1,1);
  double *d_a, *h_a;
//  int N = 1024*1024;
  hipMalloc(&d_a,Nthreads*sizeof(double));
  h_a = (double *) malloc(Nthreads*sizeof(double));
  //calculate secret key with cuda
  double startTime = clock();
  makeKernal<<<G,B>>>(N,a,b,c);
  findSecretKey<<<G,B>>>(n,p,g,h,x);
  hipDeviceSynchronize();
  hipMemcpy(h_a,d_a,Nre*Nim*sizeof(float),hipMemcpyDeviceToHost);
  ElGamalDecrypt(Zmessage,a,numOfCypher,p,x);
  free(h_a);
  hipFree(d_a);
  //done with cuda
  double endTime = clock();
  double totalTime = (endTIme-startTime)/CLOCKS_PER_SEC;
  double work = (double) p;
  double throughput = work/totalTime;
  printf("Searching all keys took %g seconds, throughput was %g values per second. \n", totalTime, throughput);
  int bufferSize = 1024;
  unsigned char *message = (unsigned char *) malloc(bufferSize*sizeof(unsigned char));
  unsigned int charsPerInt = ((n-1)/8)*numOfCypher;
  convertZToString(Zmessage, numOfCypher, message, charsPerInt);
  printf("Decrypted message = \s\"\n", message);
  printf("\n");
  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */

  return 0;
}
